#include "hip/hip_runtime.h"
//This file defines the main functions of the simulation. These functions are called in the "main" function inside the "main.cu" file.

#define _USE_MATH_DEFINES

#include "particle_positions.cuh"
#include "utilities.cuh"
#include "VTK.cuh"
#include "hashing.cuh"
#include "particle_parameters.cuh"
#include <math.h>
#include <future> 
#include <chrono>
#include <math.h>
#include "hip/hip_vector_types.h"

//declaration of all global variables that are going to be used in this file by all functions

char main_path[1024]; //stores the main path of the result instance
char vtk_group_path[1024]; //stores the path of the vtk group file
char vtu_fullpath[1024]; //stores the path of the current iteration file
char vtu_path[1024]; //stores the path of the vtu directory (where the vtu files are saved)

std::string pointDataNames[] = { "density", "pressure" }; //stores the names of the point data to display in Paraview
std::string vectorDataNames[] = {"velocity", "pressure force", "viscosity force", "st force" }; //stores the names of the vector data to display in Paraview

//initial conditions
float PARTICLE_RADIUS; //stores the particle radius value
float MASS_calc; //stores the calculated mass value
float USER_MASS; //stores the mass defined by the user in
float PARTICLE_DIAMETER; //stores the particle diameter value
float F_INITIAL_POSITION[3]; //fluid particles initial position
float F_FINAL_POSITION[3]; //fluid particles final position
float B_INITIAL_POSITION[3]; //boundary particles final position
float B_FINAL_POSITION[3]; //boundary particles final position
float V_INITIAL[3]; //initial velocity defined by the user



//controlling iteration number and simulation time
int iteration = 1; //iteration counter
float simulation_time; //in seconds
float final_time; //in seconds

//CUDA variables
int block_size;
int grid_size;

//PCISPH variables
float3* d_POSITION; //stores the pointer to the position data in the GPU
float3* d_PRED_POSITION; //stores the pointer to the predicted position data in the GPU
float3* d_VELOCITY; //stores the pointer to the velocity data in the GPU
float3* d_PRED_VELOCITY; //stores the pointer to the predicted data in the GPU
float3* d_ST_FORCE; //stores the pointer to the surface tension force data in the GPU
float3* d_VISCOSITY_FORCE; //stores the pointer to the viscosity force data in the GPU
float3* d_PRESSURE_FORCE; //stores the pointer to the pressure force data in the GPU
float3* d_NORMAL; //stores the pointer to the normal data in the GPU
float* DENSITY; //stores the pointer to the density data in the CPU
float* d_DENSITY; //stores the pointer to the density data in the GPU
float* d_PRESSURE; //stores the pointer to the pressure data in the GPU
float* d_MASS; //stores the pointer to the mass data in the GPU
int* d_TYPE; //stores the pointer to the type data in the GPU
int* d_hashtable; //stores the pointer to the hashtable data in the GPU

float vol_comp_perc; //user defined volume compression rate <- defined in section 3.3 of [2]
float dens_fluc_perc; //user defined density fluctuation rate <- defined in section 3.3 of [2]
float* d_max_force; // GPU pointer to max_force variable
float* d_max_velocity; // GPU pointer to max_velocity variable
float* d_max_rho_err; // GPU pointer to max_rho_err variable (max density error)
float* d_sum_rho_err; // GPU pointer to sum_rho_err variable (sum of all density errors across all variables to compute mean density error)
float delta_t; // time step
float max_vol_comp; // variable to stored computed value of max volume compression ( = params.rho_0 * vol_comp_perc / 100 )
float max_rho_fluc; // variable to stored computed value of max density fluctuation ( = params.rho_0 * dens_fluc_perc / 100 )
float BOUNDARY_DIAMETER; // diameter of boundary particles
float BOUNDARY_RADIUS; // radius of boundary particles
float pressure_delta; // defined in section 2.3 of [1] -> here this value is calculated without the "beta" variable, which is calculated afterwards
float max_rho_err_t_1 = 0.f; // max density error in the previous time_step
float max_rho_err = 0.f; // max density error in the current time_step (CPU memory)
bool write_pvd = true; // this tells either the program should or not write a file
char* user_results_folder = new char[256]; // user defined results folder
float save_steps; // user defined time steps to save a file

SimParams params; //host parameters

// this function reads all files in the /props folder and stores the values in the designated variables.
// If any new variable should be added or deleted in any of the props files, this function must be edited.
int fileReader() {

	//allocating memory

	char* row = new char[256]; //buffer for rows
	int row_buff_index = 0; //index for row buffer
	char* num_buffer = new char[256]; //buffer for numbers
	int num_buffer_index = 0; //index for number buffer
	float num; //stores a float variable
	float3 vec; //stores a float3 variable

	//Storing the names of varibles as they are in the files in /props folder

	char* phys_props_names[] = { "rho_0","visc_const","surface_tension_const","collision_dumping_coeff" };
	char* init_cond_names[] = {"particle_radius","mass","fluid_initial_coord","fluid_final_coord","boundary_initial_coord","boundary_final_coord","fluid_initial_velocity","maximum_volume_compression","maximum_density_fluctuation"};
	char* system_names[] = { "initial_delta_t","initial_time","final_time","neighbors_per_particle", "save_steps","results_folder"};
	
	int phys_props_size = sizeof(phys_props_names) / 8; 
	int init_cond_size = sizeof(init_cond_names) / 8;
	int system_size = sizeof(system_names) / 8;

	//storing the paths for each file

	char* phys_props_path = "./props/physical_props.txt";
	char* initial_conditions_path = "./props/initial_conditions.txt";
	char* system_path = "./props/system.txt";

	//Checking either the files exist or not -> give error and stops execution in case of error

	if (fileExists(phys_props_path) != 0) {
		std::cout << "\nERROR! Could not find physical properties file at " << phys_props_path << "\n";
		return 1;
	}

	if (fileExists(phys_props_path) != 0) {
		std::cout << "\nERROR! Could not find initial conditions file at " << phys_props_path << "\n";
		return 1;
	}

	if (fileExists(phys_props_path) != 0) {
		std::cout << "\nERROR! Could not find system names file at " << phys_props_path << "\n";
		return 1;
	}

	//reading physical properties
	std::ifstream phys_props (phys_props_path);

	for (char write2line; phys_props.get(write2line);) {
		if (phys_props.eof()) {
			break;
		}

		if (write2line == 10) {

			int i = 0;

			for (i; i < phys_props_size; i++) {
				if (strstr(row, phys_props_names[i]) != nullptr) {
					break;
				}
			}
			if (i < phys_props_size) {
				bool save_char = false;
				for (int j = 0; j < strlen(row); j++) {
					if (row[j] == 61) {
						save_char = true;
						for (int k = j; k < strlen(row); k++) {
							if (!isdigit(row[k + 1])) {
								j++;
							}
							else { break; }
						}
					}
					else if (row[j] == 59) {
						num = (float)atof(num_buffer);
						num_buffer_index = 0;
						num_buffer = new char[256];
						break;
					}
					else if ((isdigit(row[j]) || row[j] == 46 || row[j] == 45) && save_char) {
						num_buffer[num_buffer_index] = row[j];
						num_buffer_index++;
					}

				}

				if (i == 0) {
					params.rho_0 = num;
				}
				else if (i == 1) {
					params.visc_const = num;
				}
				else if (i == 2) {
					params.st_const = num;
				}
				else if (i == 3) {
					params.epsilon = num;
				}
			}
			row = new char[256];
			row_buff_index = 0;
		}
		else if (write2line != 10) {
			row[row_buff_index] = write2line;
			row_buff_index++;
		}



	}

	row = new char[256];
	row_buff_index = 0;
	phys_props.close();

	//reading initial conditions
	std::ifstream init_conds(initial_conditions_path);
	
	for (char write2line; init_conds.get(write2line);) {
		if (init_conds.eof()) {
			break;
		}

		if (write2line == 10) {

			int i = 0;

			for (i; i < init_cond_size; i++) {
				if (strstr(row, init_cond_names[i]) != nullptr) {
					break;
				}
			}
			if (i < init_cond_size) {
				if (strstr(row, "[") != nullptr) {
					bool save_char = false;
					int axis_count = 0;
					for (int j = 0; j < strlen(row); j++) {
						if (axis_count > 2) {
							axis_count = 0;
							break;
						}
						if (row[j] == 91) {
							save_char = true;
							for (int k = j; k < strlen(row); k++) {
								if (!isdigit(row[k + 1])) {
									j++;
								}
								else { break; }
							}
						}
						else if (row[j] == 44 || row[j] == 93) {
							num = (float)atof(num_buffer);
							if (axis_count == 0) {
								vec.x = num;
							} else if (axis_count == 1) {
								vec.y = num;
							}
							else if (axis_count == 2) {
								vec.z = num;
							}
							axis_count++;

							if (row[j] == 32) { 
								j++; 
								
							}

							num_buffer_index = 0;
							num_buffer = new char[256];
						}
						else if ((isdigit(row[j]) || row[j] == 46 || row[j] == 45) && save_char) {
							num_buffer[num_buffer_index] = row[j];
							num_buffer_index++;
						}
					}
				}
				else {
					bool save_char = false;
					for (int j = 0; j < strlen(row); j++) {
						if (row[j] == 61) {
							save_char = true;
							for (int k = j; k < strlen(row); k++) {
								if (!isdigit(row[k + 1])) {
									j++;
								}
								else { break; }
							}
						}
						else if (row[j] == 59) {
							num = (float)atof(num_buffer);
							num_buffer_index = 0;
							num_buffer = new char[256];
							break;
						}
						else if ((isdigit(row[j]) || row[j] == 46 || row[j] == 45) && save_char) {
							num_buffer[num_buffer_index] = row[j];
							num_buffer_index++;
						}

					}
				}


				if (i == 0) {
					PARTICLE_RADIUS = num;
				}
				else if (i == 1) {
					USER_MASS = num;
				}
				else if (i == 2) {
					F_INITIAL_POSITION[0] = vec.x;
					F_INITIAL_POSITION[1] = vec.y;
					F_INITIAL_POSITION[2] = vec.z;
				}
				else if (i == 3) {
					F_FINAL_POSITION[0] = vec.x;
					F_FINAL_POSITION[1] = vec.y;
					F_FINAL_POSITION[2] = vec.z;
				}
				else if (i == 4) {
					B_INITIAL_POSITION[0] = vec.x;
					B_INITIAL_POSITION[1] = vec.y;
					B_INITIAL_POSITION[2] = vec.z;
				}
				else if (i == 5) {
					B_FINAL_POSITION[0] = vec.x;
					B_FINAL_POSITION[1] = vec.y;
					B_FINAL_POSITION[2] = vec.z;
				}
				else if (i == 6) {
					V_INITIAL[0] = vec.x;
					V_INITIAL[1] = vec.y;
					V_INITIAL[2] = vec.z;
				}
				else if (i == 7) {
					vol_comp_perc = num;
				}
				else if (i == 8) {
					dens_fluc_perc = num;
				}
			}
			row = new char[256];
			row_buff_index = 0;
		}
		else if (write2line != 10) {
			row[row_buff_index] = write2line;
			row_buff_index++;
		}



	}

	row = new char[256];
	row_buff_index = 0;
	init_conds.close();

	std::ifstream system_vars(system_path);

	for (char write2line; system_vars.get(write2line);) {
		if (system_vars.eof()) {
			break;
		}

		if (write2line == 10) {

			int i = 0;

			for (i; i < system_size; i++) {
				if (strstr(row, system_names[i]) != nullptr) {
					break;
				}
			}
			if (i < system_size) {
				bool save_char = false;
				if (strstr(row, "\"") != nullptr) {
					for (int j = 0; j < strlen(row); j++) {
						if (row[j] == 34 && !save_char) {
							save_char = true;
							for (int k = j; k < strlen(row); k++) {
								if (row[k+1] == 32) {
									j++;
								}
								else { break; }
							}
						}
						else if (row[j] == 34 && save_char) {
							break;
						}
						else if (save_char){
							num_buffer[num_buffer_index] = row[j];
							num_buffer_index++;
						}

					}
				}
				else {
					for (int j = 0; j < strlen(row); j++) {
						if (row[j] == 61) {
							save_char = true;
							for (int k = j; k < strlen(row); k++) {
								if (!isdigit(row[k + 1])) {
									j++;
								}
								else { break; }
							}
						}
						else if (row[j] == 59) {
							num = (float)atof(num_buffer);
							num_buffer_index = 0;
							num_buffer = new char[256];
							break;
						}
						else if ((isdigit(row[j]) || row[j] == 46 || row[j] == 45) && save_char) {
							num_buffer[num_buffer_index] = row[j];
							num_buffer_index++;
						}

					}
				}
				

				if (i == 0) {
					delta_t = num;
				}
				else if (i == 1) {
					simulation_time = num;
				}
				else if (i == 2) {
					final_time = num;
				}
				else if (i == 3) {
					params.particles_per_row = (int)num;
				}
				else if (i == 4) {
					save_steps = num;
				}
				else if (i == 5) {
					user_results_folder = num_buffer;
				}

			}
			row = new char[256];
			row_buff_index = 0;
		}
		else if (write2line != 10) {
			row[row_buff_index] = write2line;
			row_buff_index++;
		}

	}



	return 0;
}

// this function initialized the execution. It creates the particles, calculates some variables and allocate memory in the GPU for the main loop.
int initialize() {
	
	//Display GPU information and checking if the program is running in a CUDA capable machine or not.

	hipDeviceProp_t* prop = new hipDeviceProp_t;
	gpuErrchk(hipGetDeviceProperties(prop,0));
	std::cout << "-----------------------------------------------\n";
	std::cout << "DEVICE PROPERTIES:\n" << "Device name: " << prop->name << "\n" <<
		"Max number of threads per block: " << prop->maxThreadsPerBlock << "\n" <<
		"Total global memory: " << dround(prop->totalGlobalMem/1e9,2) << " gigabytes\n" <<
		"Registers per block: " << prop->regsPerBlock << "\n" << 
		"Shared Memory per block: " << prop->sharedMemPerBlock << " bytes\n" <<
		"-----------------------------------------------\n";

	block_size = prop->maxThreadsPerBlock; //stores the size of the thread blocks. Here its set to be the same size of the max threads per block of your GPU (1024 in the modern devices).

	max_vol_comp = params.rho_0 * vol_comp_perc / 100; 
	max_rho_fluc = params.rho_0 * dens_fluc_perc / 100;

	//If the user did not define a mass, calculate it.

	if (USER_MASS == 0) {
		MASS_calc = params.rho_0 * (float)M_PI * pow(PARTICLE_RADIUS, 3.f) / 3.f * 4.f;
	}
	else {
		MASS_calc = USER_MASS;
	}

	PARTICLE_DIAMETER = 2 * PARTICLE_RADIUS;

	// get main path of simulation
	getMainPath(main_path);

	// write path for vtu files
	strcpy(vtu_path, main_path);
	strcat(vtu_path, "/vtu");

	// write path for vtk group file
	strcpy(vtk_group_path, main_path);
	strcat(vtk_group_path, "/PCISPH.pvd");

	// create directory for vtu files
	CreateDir(vtu_path);

	float VOLUME = 1;
	const int SIMULATION_DIMENSION = 3; //3 for a 3D simulation

	// Get number per dimension (NPD) of FLUID particles for hexadecimal packing (assuming use of makeprism function)
	
	int NPD[3]; //Number per dimension

	for (int i = 0; i < 3; i++) {
		if (i == 1) {
			NPD[i] = static_cast<int>(floor((F_FINAL_POSITION[i] - F_INITIAL_POSITION[i]) / (sqrt(3.f) / 2.f * PARTICLE_DIAMETER)));
			VOLUME = VOLUME * (F_FINAL_POSITION[i] - F_INITIAL_POSITION[i]);
		}
		else {
			NPD[i] = static_cast<int>(floor((F_FINAL_POSITION[i] - F_INITIAL_POSITION[i]) / PARTICLE_DIAMETER));
			VOLUME = VOLUME * (F_FINAL_POSITION[i] - F_INITIAL_POSITION[i]);
		}
	}

	//Writing NPD to device
	int* D_NPD; //Device pointer to NPD variable
	gpuErrchk(hipMalloc((void**)&D_NPD, SIMULATION_DIMENSION * sizeof(float))); //Allocate GPU memory
	gpuErrchk(hipMemcpy(D_NPD, NPD, SIMULATION_DIMENSION * sizeof(float), hipMemcpyHostToDevice)); //Write NPD to D_NPD

	params.N = NPD[0] * NPD[1] * NPD[2]; //number of fluid particles
	int SIM_SIZE = params.N * SIMULATION_DIMENSION; //size of the fluid part of the simulation
	const int x = 40; // Number of particles inside the smoothing length
	params.h = powf(3.f * VOLUME * x / (4.f * (float)M_PI * params.N), 1.f / 3.f); //smoothing length
	params.invh = 1 / params.h; // inverse of smoothing length (this is calculated to make things faster in the main loop)

	float3 f_initial; //initial position taking in account the offset of the particle radius
	f_initial.x = F_INITIAL_POSITION[0] + PARTICLE_RADIUS; 
	f_initial.y = F_INITIAL_POSITION[1] + PARTICLE_RADIUS;
	f_initial.z = F_INITIAL_POSITION[2] + PARTICLE_RADIUS;

	size_t bytes_fluid_particles = SIM_SIZE * sizeof(float);

	float3* FLUID_POSITIONS; //host pointer (CPU memory)
	FLUID_POSITIONS = (float3*)malloc(bytes_fluid_particles); //allocating CPU memory

	float3* D_FLUID_POSITIONS; //device pointer (GPU memory)
	gpuErrchk(hipMalloc((void**)&D_FLUID_POSITIONS, bytes_fluid_particles)); //allocating GPU memory

	// grid -> number of blocks
	// block -> number of threads

	grid_size = params.N / block_size + 1; //defining number of blocks

	//generate locations for each particle
	//check "particle_positions.cuh" file in /lib folder for more details
	makePrism << <grid_size, block_size >> > (D_FLUID_POSITIONS, PARTICLE_DIAMETER, f_initial, D_NPD, params.N);

	BOUNDARY_DIAMETER = params.h/2; //defining the diameter of a boundary particle as stated in section 3.2 in [2]
	BOUNDARY_RADIUS = params.h/4;

	// Get number per dimension (NPD) of BOUNDARY particles without compact packing (assuming use of makebox function)
	for (int i = 0; i < 3; i++) {
		NPD[i] = static_cast<int>(ceil((B_FINAL_POSITION[i] - B_INITIAL_POSITION[i]) / BOUNDARY_DIAMETER)) + 2;

	}

	params.B = NPD[0] * NPD[1] * NPD[2] - (NPD[0] - 2) * (NPD[1] - 2) * (NPD[2] - 2); //Number of boundary particles
	SIM_SIZE = NPD[0] * NPD[1] * NPD[2] * SIMULATION_DIMENSION;

	float3 b_initial; //initial position taking in account the offset of the boundary particle radius
	b_initial.x = B_INITIAL_POSITION[0] - BOUNDARY_RADIUS;
	b_initial.y = B_INITIAL_POSITION[1] - BOUNDARY_RADIUS;
	b_initial.z = B_INITIAL_POSITION[2] - BOUNDARY_RADIUS;
	float3 b_final; //final position taking in account the offset of the boundary particle radius
	b_final.x = b_initial.x + BOUNDARY_DIAMETER * (NPD[0] - 1);
	b_final.y = b_initial.y + BOUNDARY_DIAMETER * (NPD[1] - 1);
	b_final.z = b_initial.z + BOUNDARY_DIAMETER * (NPD[2] - 1);

	size_t bytes_boundary_particles = SIM_SIZE * sizeof(float); // number of bytes the boundary particles are occupying
	float3* BOUNDARY_POSITIONS; //host pointer (CPU memory)
	BOUNDARY_POSITIONS = (float3*)malloc(bytes_boundary_particles); //allocate memory in the host

	float3* D_BOUNDARY_POSITIONS; //device pointer (GPU memory)
	gpuErrchk(hipMalloc((void**)&D_BOUNDARY_POSITIONS, bytes_boundary_particles)); //allocate memory in the device

	// this function makes an empty box with walls with 1 particle of thickness
	// check "particle_positions.cuh" file in /lib folder for more details
	makeBox(D_BOUNDARY_POSITIONS, BOUNDARY_DIAMETER, b_initial, b_final, block_size, D_NPD,NPD, SIMULATION_DIMENSION);

	params.T = params.N + params.B; //Total number of particles

	//writing particle position memory from GPU to CPU (note the "hipMemcpyDeviceToHost" statement in the functions below)
	gpuErrchk(hipMemcpy(FLUID_POSITIONS, D_FLUID_POSITIONS, bytes_fluid_particles, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(BOUNDARY_POSITIONS, D_BOUNDARY_POSITIONS, bytes_boundary_particles, hipMemcpyDeviceToHost));

	// Free GPU memory for fluid particles (this memory will be reallocated with another name soon)
	hipFree(D_FLUID_POSITIONS);

	// Getting hashtable size
	uint tmp_elev = 19;
	params.hashtable_size = unsignedIntPow(2, tmp_elev);
	while (params.T > params.hashtable_size) {
		tmp_elev++;
		params.hashtable_size = unsignedIntPow(2, tmp_elev);
	}

	// Transfering params to GPU
	
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(&d_params), &params, sizeof(SimParams)))

	int* hashtable = new int[params.hashtable_size * params.particles_per_row];

	//this loop creates an empty hashtable (full of -1s)
	for (uint i = 0; i < params.hashtable_size; ++i) {
		for (uint j = 0; j < params.particles_per_row; j++) {
			hashtable[i * params.particles_per_row + j] = -1;
		}
	}

	//allocating 2D memory for hashtable 
	gpuErrchk(hipMallocPitch(&params.d_hashtable, &params.pitch, params.particles_per_row * sizeof(int), params.hashtable_size));
	//writing clean hashtable to GPU
	gpuErrchk(hipMemcpy2D(params.d_hashtable, params.pitch, hashtable, params.particles_per_row * sizeof(int), params.particles_per_row * sizeof(int), params.hashtable_size, hipMemcpyHostToDevice));

	grid_size = params.B / block_size + 1;
	//this function makes a functional hashtable
	hashParticlePositions << <grid_size, block_size >> > (D_BOUNDARY_POSITIONS, params.B);
	gpuErrchk(hipPeekAtLastError()); // this is for checking if there was any error during the kernel execution
	gpuErrchk(hipDeviceSynchronize());
	float* d_boundary_mass; //pointer to device memory of boundary "fake" mass ( or psi )
	gpuErrchk(hipMalloc((void**)&d_boundary_mass, params.B * sizeof(float)));

	// calculates "fake" mass (or psi) for each boundary particle as state in [3]
	// check "particle_parameters.cuh" file in /lib folder for more details
	boundaryPsi << <grid_size, block_size >> > (d_boundary_mass,D_BOUNDARY_POSITIONS);

	float* boundary_mass = (float*)malloc(params.B * sizeof(float)); //CPU pointer to boundary mass
	//copy boundary mass from GPU to CPU
	gpuErrchk(hipMemcpy(boundary_mass, d_boundary_mass, (size_t)params.B * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_boundary_mass));

	float3* d_boundary_normal; //device pointer for boundary normal
	gpuErrchk(hipMalloc((void**)&d_boundary_normal, params.B * 3 * sizeof(float)));

	// calculate normal for boundary particles
	// check "particle_parameters.cuh" file in /lib folder for more details
	boundaryNormal << <grid_size, block_size >> > (d_boundary_normal, D_BOUNDARY_POSITIONS, b_initial, b_final);

	float3* boundary_normal = (float3*)malloc(params.B * sizeof(float3)); //pointer for CPU memory of boundary normal
	// copying boundary normal memory from GPU to CPU
	gpuErrchk(hipMemcpy(boundary_normal, d_boundary_normal, (size_t)params.B * 3 * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_boundary_normal)); //cleaning GPU memory of boundary normal (this will be reallocated later with another name)

	// writing boundary vtu file

	float** boundary_point_data[] = { &boundary_mass };
	int size_pointData = sizeof(boundary_point_data) / sizeof(double);
	float3** boundary_vectorData[] = { &boundary_normal };
	int size_vectorData = sizeof(boundary_vectorData) / sizeof(double);

	std::string boundary_pointDataNames[] = { "psi" };
	std::string boundary_vectorDataNames[] = { "normal" };

	VTU_Writer(main_path, iteration, BOUNDARY_POSITIONS, params.B, boundary_point_data, boundary_vectorData, boundary_pointDataNames, boundary_vectorDataNames, size_pointData, size_vectorData, vtu_fullpath, 1);

	hipFree(params.d_hashtable); //cleaning GPU from hashtable memory

	hipFree(D_BOUNDARY_POSITIONS); //cleaning GPU from boundary particle memory

	// calculating pressure delta (without the beta variable) as stated in section 2.3 of [1]

	int count = 0;
	float min_r = std::numeric_limits<float>::infinity();
	int selected_index;
	int tmp_size = static_cast<int>(ceil((2 * (params.h + PARTICLE_DIAMETER)) / PARTICLE_DIAMETER));
	float3* tmp_points = (float3*)malloc(tmp_size * tmp_size * tmp_size * 3 * sizeof(float));

	// generating fake particle positions without any packing method (the same is done in [5])
	for (float i = -params.h - PARTICLE_DIAMETER; i <= params.h + PARTICLE_DIAMETER; i += PARTICLE_DIAMETER) {
		for (float j = -params.h - PARTICLE_DIAMETER; j <= params.h + PARTICLE_DIAMETER; j += PARTICLE_DIAMETER) {
			for (float k = -params.h - PARTICLE_DIAMETER; k <= params.h + PARTICLE_DIAMETER; k += PARTICLE_DIAMETER) {
				tmp_points[count].x = i;
				tmp_points[count].y = j;
				tmp_points[count].z = k;
				count++;
				float r = sqrt(i*i+j*j+k*k);
				if (r < min_r) {
					min_r = r;
					selected_index = count;
				}
			}
		}
	}

	float3 selected_point = tmp_points[selected_index];
	float3 r_vector;
	float r;
	float3 Grad_W;
	Grad_W.x = 0.f;
	Grad_W.y = 0.f;
	Grad_W.z = 0.f;
	float dot_Grad_W = 0.f;

	// summation of the calculated kernel gradients
	for (int i = 0; i < count; i++) {
		r_vector.x = tmp_points[i].x - selected_point.x;
		r_vector.y = tmp_points[i].y - selected_point.y;
		r_vector.z = tmp_points[i].z - selected_point.z;
		r = sqrt(r_vector.x* r_vector.x + r_vector.y* r_vector.y + r_vector.z* r_vector.z);

		if (r <= params.h) {
			float3 inst_Grad_W = Poly6_Gradient(selected_index, i, tmp_points, r, params.h, params.invh);

			Grad_W.x += inst_Grad_W.x;
			Grad_W.y += inst_Grad_W.y;
			Grad_W.z += inst_Grad_W.z;

			dot_Grad_W += dot_product(inst_Grad_W, inst_Grad_W);
		}

	}

	pressure_delta = -dot_product(Grad_W, Grad_W) - dot_Grad_W;

	//Initializing main particle variables

	//Defining and allocating main position variable
	
	float3* POSITION = (float3*)malloc(params.T*sizeof(float3));
	for (uint i = 0; i < params.N; i++) {
		POSITION[i].x = FLUID_POSITIONS[i].x;
		POSITION[i].y = FLUID_POSITIONS[i].y;
		POSITION[i].z = FLUID_POSITIONS[i].z;
	}

	for (uint i = params.N; i < params.T; i++) {
		POSITION[i].x = BOUNDARY_POSITIONS[i - params.N].x;
		POSITION[i].y = BOUNDARY_POSITIONS[i - params.N].y;
		POSITION[i].z = BOUNDARY_POSITIONS[i - params.N].z;
	}

	free(BOUNDARY_POSITIONS);
	free(FLUID_POSITIONS);

	
	gpuErrchk(hipMalloc((void**)&d_POSITION, 3*params.T*sizeof(float)));
	gpuErrchk(hipMemcpy(d_POSITION, POSITION, 3*params.T*sizeof(float), hipMemcpyHostToDevice));

	//Allocating memory for predicted positions and copying previous position vectors
	gpuErrchk(hipMalloc((void**)&d_PRED_POSITION, 3 * params.T * sizeof(float)));
	gpuErrchk(hipMemcpy(d_PRED_POSITION, POSITION, 3 * params.T * sizeof(float), hipMemcpyHostToDevice));

	//Allocating memory for predicted velocity
	gpuErrchk(hipMalloc((void**)&d_PRED_VELOCITY, 3 * params.N * sizeof(float)));

	//Defining and allocating main velocity variable
	
	float3* VELOCITY = (float3*)malloc(params.N*sizeof(float3));
	for (uint i = 0; i < params.N; i++) {
		VELOCITY[i].x = V_INITIAL[0];
		VELOCITY[i].y = V_INITIAL[1];
		VELOCITY[i].z = V_INITIAL[2];
	}

	gpuErrchk(hipMalloc((void**)&params.d_VELOCITY,  params.N*sizeof(float3)));
	gpuErrchk(hipMemcpy(params.d_VELOCITY, VELOCITY,  params.N*sizeof(float3), hipMemcpyHostToDevice));

	//Defining and allocating main st force variable
	
	float3* ST_FORCE = (float3*)malloc( params.N*sizeof(float3));
	for (uint i = 0; i < params.N; i++) {
		ST_FORCE[i].x = 0.f;
		ST_FORCE[i].y = 0.f;
		ST_FORCE[i].z = 0.f;
	}

	gpuErrchk(hipMalloc((void**)&params.d_ST_FORCE,  params.N*sizeof(float3)));

	//Defining and allocating main viscosity force variable
	float3* VISCOSITY_FORCE = (float3*)malloc( params.N*sizeof(float3));
	for (uint i = 0; i < params.N; i++) {
		VISCOSITY_FORCE[i].x = 0.f;
		VISCOSITY_FORCE[i].y = 0.f;
		VISCOSITY_FORCE[i].z = 0.f;
	}

	gpuErrchk(hipMalloc((void**)&params.d_VISCOSITY_FORCE,  params.N*sizeof(float3)));

	//Defining and allocating main pressure force variable
	float3* PRESSURE_FORCE = (float3*)malloc( params.N*sizeof(float3));
	for (uint i = 0; i < params.N; i++) {
		PRESSURE_FORCE[i].x = 0.f;
		PRESSURE_FORCE[i].y = 0.f;
		PRESSURE_FORCE[i].z = 0.f;
	}

	gpuErrchk(hipMalloc((void**)&params.d_PRESSURE_FORCE,  params.N*sizeof(float3)));

	//Defining and allocating main normal variable
	float3* NORMAL = (float3*)malloc(3*params.T*sizeof(float3));
	for (uint i = 0; i < params.N; i++) {
		NORMAL[i].x = 0.f;
		NORMAL[i].y = 0.f;
		NORMAL[i].z = 0.f;
	}	

	for (uint i = params.N; i < params.T; i++) {
		NORMAL[i].x = boundary_normal[i - params.N].x;
		NORMAL[i].y = boundary_normal[i - params.N].y;
		NORMAL[i].z = boundary_normal[i - params.N].z;
	}

	free(boundary_normal);

	gpuErrchk(hipMalloc((void**)&params.d_NORMAL, params.T*sizeof(float3)));
	gpuErrchk(hipMemcpy(params.d_NORMAL, NORMAL, params.T*sizeof(float3), hipMemcpyHostToDevice));

	//Defining and allocating main density array
	float* DENSITY = (float*)malloc(params.N * sizeof(float));
	for (uint i = 0; i < params.N; i++) {
		DENSITY[i] = 0.f;
	}

	gpuErrchk(hipMalloc((void**)&params.d_DENSITY, params.N * sizeof(float)));
	gpuErrchk(hipMemcpy(params.d_DENSITY, DENSITY, params.N * sizeof(float), hipMemcpyHostToDevice));

	//Defining and allocating main pressure array
	float* PRESSURE = (float*)malloc(params.N * sizeof(float));
	for (uint i = 0; i < params.N; i++) {
		PRESSURE[i] = 0;
	}

	gpuErrchk(hipMalloc((void**)&params.d_PRESSURE, params.N * sizeof(float)));

	//Defining and allocating main mass array
	
	float* MASS = (float*)malloc(params.T * sizeof(float));
	for (uint i = 0; i < params.N; i++) {
		MASS[i] = MASS_calc;
	}

	for (uint i = params.N; i < params.T; i++) {
		MASS[i] = boundary_mass[i - params.N];
	}

	free(boundary_mass);
	
	gpuErrchk(hipMalloc((void**)&params.d_MASS, params.T * sizeof(float)));
	gpuErrchk(hipMemcpy(params.d_MASS, MASS, params.T * sizeof(float), hipMemcpyHostToDevice));

	//Defining and allocating main type array (0 if fluid, 1 if boundary)
	int* TYPE = (int*)malloc(params.T * sizeof(int));
	for (uint i = 0; i < params.N; i++) {
		TYPE[i] = 0;
	}

	for (uint i = params.N; i < params.T; i++) {
		TYPE[i] = 1;
	}

	gpuErrchk(hipMalloc((void**)&params.d_TYPE, params.T * sizeof(int)));
	gpuErrchk(hipMemcpy(params.d_TYPE, TYPE, params.T * sizeof(int), hipMemcpyHostToDevice));

	//Defining and allocating memory to store max density error
	gpuErrchk(hipMalloc((void**)&d_max_rho_err, sizeof(float)));

	//Defining and allocating memory to store max force value
	gpuErrchk(hipMalloc((void**)&d_max_force, sizeof(float)));

	//Defining and allocating memory to store max velocity value
	gpuErrchk(hipMalloc((void**)&d_max_velocity, sizeof(float)));

	//Defining and allocating memory to store summation of density errors to calculate average error
	gpuErrchk(hipMalloc((void**)&d_sum_rho_err, sizeof(float)));

	//defining gravity vector
	params.gravity.x = 0.f;
	params.gravity.y = -9.81f;
	params.gravity.z = 0.f;

	//Defining variables to write VTU files
	float** pointData[] = { &DENSITY, &PRESSURE }; // here the CPU pointers to the FLOAT variables that you want to write in the VTU must be defined
	float3** vectorData[] = { &VELOCITY, &PRESSURE_FORCE, &VISCOSITY_FORCE, &ST_FORCE  }; // here the CPU pointers to the VEC3D variables that you want to write in the VTU must be defined

	size_pointData = sizeof(pointData) / 8;
	size_vectorData = sizeof(vectorData) / 8;

	VTU_Writer(vtu_path, iteration, POSITION, params.N, pointData, vectorData, pointDataNames, vectorDataNames, size_pointData, size_vectorData, vtu_fullpath);

	VTK_Group(vtk_group_path, vtu_fullpath, simulation_time);

	// Initialize main hashtable

	//allocating memory for GPU hashtable
	gpuErrchk(hipMallocPitch(&params.d_hashtable, &params.pitch, params.particles_per_row * sizeof(int), params.hashtable_size));

	writeTimeKeeper(main_path,max_rho_err); //time keeper file with values for time, iteration and max density error

	hipMemcpyToSymbol(HIP_SYMBOL(&d_params),&params,sizeof(SimParams),hipMemcpyHostToDevice);

	std::cout << params.N << " Fluid particles\n"
		<< params.B << " Boundary particles\n"
		<< "Total of " << params.T << " particles.\n"
		<< "Smoothing radius = " << params.h << " m.\n"
		<< "hashtable size = " << params.hashtable_size << "\n";

	gpuErrchk(hipPeekAtLastError()); // this is for checking if there was any error during the kernel execution
	gpuErrchk(hipDeviceSynchronize());

	return 0;
}

// here is where the magic happens
// comments with -> refer to the same lines of the pseudo code in Algorithm 2 in [2]
// -> while animating do
int mainLoop() {
	
	// -> for each particle i,b do
	//	-> find neighbors Ni,b(t)

	// here the hashtable is initialized and reset
	grid_size = params.hashtable_size / block_size + 1;
	hashtableReset << <grid_size, block_size >> >  ();
	
	// then a new hashtable is created
	grid_size = params.T / block_size + 1;
	hashParticlePositions << <grid_size, block_size >> > ();

	// -> for each particle i do

	// here there are tow more step than the pseudo algorithm:
	
	// calculate density
	grid_size = params.N / block_size + 1;
	DensityCalc << <grid_size, block_size >> > ();

	// and the normal for each fluid particle
	
	fluidNormal << <grid_size, block_size >> > ();
	
	// -> compute forces Fi for viscosity and surface tension (gravity is only accounted later)
	nonPressureForces << <grid_size, block_size >> > ();
	
	// -> set pressure pi(t) = 0 

	resetPressure << <grid_size, block_size >> > ();
	// here the step to set the pressure force value as 0 is ignored as it is done on later steps

	// calculate the pressure coefficient as in Equation 8 of [1]
	float pressure_coeff = -1 / (2 * powf(MASS_calc * delta_t / params.rho_0, 2) * pressure_delta);
	
	gpuErrchk(hipPeekAtLastError()); // this is for checking if there was any error during the kernel execution
	gpuErrchk(hipDeviceSynchronize()); 

	int _k_ = 0; // defined with underscores to prevent overwritting 
	// -> while k < 3 do
	while (_k_ < 3) {
		
		// -> for each particle i do
		//  -> predicit velocity 
		//  -> predicit position 
		grid_size = params.N / block_size + 1;
		positionAndVelocity << <grid_size, block_size >> > (d_params.d_PRED_POSITION,d_params.d_PRED_VELOCITY,d_params.d_POSITION, d_params.d_VELOCITY, delta_t);

		// -> predict world collision
		collisionHandler << <grid_size, block_size >> > ();
		
		// reset and create new hashtable
		grid_size = params.hashtable_size / block_size + 1;
		hashtableReset << <grid_size, block_size >> > ();
		grid_size = params.T / block_size + 1;
		hashParticlePositions << <grid_size, block_size >> > ();

		// update distances to neighbors is unnecessary here

		// -> predict density
		grid_size = params.N / block_size + 1;
		DensityCalc << <grid_size, block_size >> > ();

		// -> predict density variation and -> update pressure
		PressureCalc << <grid_size, block_size >> > (pressure_coeff);

		// -> compute pressure force

		PressureForceCalc << <grid_size, block_size >> > ();

		_k_++;
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
	}

	// -> compute new velocity and compute new position
	positionAndVelocity << <grid_size, block_size >> > (d_params.d_POSITION, d_params.d_VELOCITY, d_params.d_POSITION, d_params.d_VELOCITY, delta_t);

	// -> compute new world collision
	collisionHandler << <grid_size, block_size >> > ();

	// -> adapt time step

	// criterias for changes in delta_t value according to session 3.3 of [2]

	// getting max velocity, max force, max density error and average density error
	max_rho_err_t_1 = max_rho_err;
	float max_velocity = 0.f;
	float max_force = 0.f;
	float sum_rho_err = 0.f;
	resetValues<<<1,1>>>(d_max_velocity, d_max_force, d_sum_rho_err, d_max_rho_err);
	grid_size = params.N / block_size + 1;
	getMaxVandF << <grid_size, block_size >> > (d_max_force, d_max_velocity, d_sum_rho_err,d_max_rho_err);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipMemcpy(&max_velocity, d_max_velocity, sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&max_force, d_max_force, sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&sum_rho_err, d_sum_rho_err, sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&max_rho_err, d_max_rho_err, sizeof(float), hipMemcpyDeviceToHost));
	
	float avg_rho_err = sum_rho_err / params.N;

	// criterias for delta_t increase

	bool criteria1 = 0.19f * sqrt(params.h / max_force) > delta_t;
	bool criteria2 = max_rho_err < 4.5f * max_vol_comp;
	bool criteria3 = avg_rho_err < 0.9f * max_vol_comp;
	bool criteria4 = 0.39f * (params.h/max_velocity) > delta_t;

	if (criteria1 && criteria2 && criteria3 && criteria4) {
		delta_t += delta_t * 0.2f / 100;
	}

	// criterias for delta_t decrease

	criteria1 = 0.2f * sqrt(params.h / max_force) < delta_t;
	criteria2 = max_rho_err > 5.5f * max_vol_comp;
	criteria3 = avg_rho_err > max_vol_comp;
	criteria4 = 0.4f * (params.h / max_velocity) <= delta_t;

	if (criteria1 || criteria2 || criteria3 || criteria4) {
		delta_t -= delta_t * 0.2f / 100;
	}

	// criterias for shock handling

	criteria1 = max_rho_err - max_rho_err_t_1 > 8 * max_vol_comp;
	criteria2 = max_rho_err > max_rho_fluc;
	criteria3 = 0.45f * (params.h/max_velocity) < delta_t;

	if (criteria1 || criteria2 || criteria3) {

		//get last iteration greater or equal to 2
		int last_iter = getLastIter(main_path);
		char* iter_path = new char[100];
		char* num_buffer = new char[32];
		while (iteration - last_iter < 2) {
			itoa(last_iter, num_buffer, 10);
			strcpy(iter_path, vtu_path);
			strcat(iter_path, "/iter");
			strcat(iter_path, num_buffer);
			strcat(iter_path, ".vtu");
			remove(iter_path);
			last_iter = getLastIter(main_path);
			num_buffer = new char[32];
			iter_path = new char[100];
		}

		std::cout << "\n\nSHOCK DETECTED! RETURNING " << iteration - last_iter << " ITERATIONS!\n" << std::endl;
		write_pvd = false;
		//SHOCK DETECTED

		delta_t -= delta_t * 0.5f;

		iteration = last_iter;
		if (iteration <= 0) {
			std::cout << "\nIMPOSSIBLE TO RETURN 2 ITERATIONS! TERMINATING SIMULATION\n" << std::endl;
			return 1;
		}

		float3* position = (float3*)malloc(params.N * sizeof(float3));
		float3* velocity = (float3*)malloc(params.N * sizeof(float3));

		itoa(iteration, num_buffer, 10);
		strcpy(iter_path, vtu_path);
		strcat(iter_path, "/iter");
		strcat(iter_path, num_buffer);
		strcat(iter_path, ".vtu");

		//read VTU file to go to the required step backwards
		readVTU(iter_path, position, velocity); 

		//get the correct time of the previous iteration 
		getNewSimTime(main_path);
		//edit PVD (group) file with the correct information
		rewritePVD(main_path);


		gpuErrchk(hipMemcpy(d_POSITION, position, 3 * params.N * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_VELOCITY, velocity, 3 * params.N * sizeof(float), hipMemcpyHostToDevice));

		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		return 0;
	}

	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	if (simulation_time + delta_t > final_time) {
		simulation_time = final_time;
	}
	else {
		simulation_time += delta_t;
	}
	
	iteration++;

	writeTimeKeeper(main_path,max_rho_err);

	return 0;
}

// This function writes VTU files using multiple CPU cores
void multiprocessor_writer() {

	char buf[1024];
	itoa(iteration, buf, 10);
	strcpy(vtu_fullpath, vtu_path);
	strcat(vtu_fullpath, "/iter");
	strcat(vtu_fullpath, buf);
	strcat(vtu_fullpath, ".vtu");

	std::future<void> write_vtu;

	float3* write_position = (float3*)malloc(params.N * sizeof(float3));
	float3* write_velocity = (float3*)malloc(params.N * sizeof(float3));
	float3* write_p_force = (float3*)malloc(params.N * sizeof(float3));
	float3* write_st_force = (float3*)malloc(params.N * sizeof(float3));
	float3* write_v_force = (float3*)malloc(params.N * sizeof(float3));
	float* write_pressure = (float*)malloc(params.N * sizeof(float));
	float* write_density = (float*)malloc(params.N * sizeof(float));

	void* d_params_gpu_pptr;
	hipGetSymbolAddress(&d_params_gpu_pptr, HIP_SYMBOL(&d_params));
	SimParams* d_params_gpu_ptr = (SimParams*)d_params_gpu_pptr;
	SimParams d_params_gpu = d_params_gpu_ptr[0];

	gpuErrchk(hipMemcpy(write_position, d_POSITION, params.N * sizeof(float3), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(write_velocity, d_VELOCITY, params.N * sizeof(float3), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(write_p_force, d_params_gpu.d_PRESSURE_FORCE, params.N * sizeof(float3), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(write_st_force, d_params_gpu.d_ST_FORCE, params.N * sizeof(float3), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(write_v_force, d_params_gpu.d_VISCOSITY_FORCE, params.N * sizeof(float3), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(write_density, d_params_gpu.d_DENSITY, params.N * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(write_pressure, d_params_gpu.d_PRESSURE, params.N * sizeof(float), hipMemcpyDeviceToHost));

	float** pointData[] = { &write_density, &write_pressure };
	float3** vectorData[] = { &write_velocity, &write_p_force, &write_v_force, &write_st_force };

	int size_pointData = sizeof(pointData) / 8;
	int size_vectorData = sizeof(vectorData) / 8;

	write_vtu = std::async(std::launch::async, VTU_Writer, vtu_path, iteration, write_position, params.N, pointData, vectorData, pointDataNames, vectorDataNames, size_pointData, size_vectorData, vtu_fullpath,2);
		
	if (write_pvd == true) {
		strcpy(buf, vtu_fullpath);
		VTK_Group(vtk_group_path, buf, simulation_time);
	}
	write_pvd = true;

	return;
}